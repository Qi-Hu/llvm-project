
#include <hip/hip_runtime.h>
// RUN: %clang -### -target x86_64-linux-gnu -nocudalib -ccc-print-bindings -fgpu-rdc \
// RUN:        --offload-new-driver --offload-arch=sm_35 --offload-arch=sm_70 %s 2>&1 \
// RUN: | FileCheck -check-prefix BINDINGS %s

//      BINDINGS: "nvptx64-nvidia-cuda" - "clang", inputs: ["[[INPUT:.+]]"], output: "[[PTX_SM_35:.+]]"
// BINDINGS-NEXT: "nvptx64-nvidia-cuda" - "NVPTX::Assembler", inputs: ["[[PTX_SM_35]]"], output: "[[CUBIN_SM_35:.+]]"
// BINDINGS-NEXT: "nvptx64-nvidia-cuda" - "clang", inputs: ["[[INPUT]]"], output: "[[PTX_SM_70:.+]]"
// BINDINGS-NEXT: "nvptx64-nvidia-cuda" - "NVPTX::Assembler", inputs: ["[[PTX_SM_70:.+]]"], output: "[[CUBIN_SM_70:.+]]"
// BINDINGS-NEXT: "x86_64-unknown-linux-gnu" - "Offload::Packager", inputs: ["[[CUBIN_SM_35]]", "[[CUBIN_SM_70]]"], output: "[[BINARY:.+]]"
// BINDINGS-NEXT: "x86_64-unknown-linux-gnu" - "clang", inputs: ["[[INPUT]]", "[[BINARY]]"], output: "[[HOST_OBJ:.+]]"
// BINDINGS-NEXT: "x86_64-unknown-linux-gnu" - "Offload::Linker", inputs: ["[[HOST_OBJ]]"], output: "a.out"

// RUN: %clang -### -target x86_64-linux-gnu -nocudalib -ccc-print-bindings -fgpu-rdc \
// RUN:        --offload-new-driver --offload-arch=sm_35 --offload-arch=sm_70 %s 2>&1 \
// RUN: | FileCheck -check-prefix BINDINGS-HOST %s

// BINDINGS-HOST: # "x86_64-unknown-linux-gnu" - "clang", inputs: ["[[INPUT:.+]]"], output: "[[OUTPUT:.+]]"
// BINDINGS-HOST: # "x86_64-unknown-linux-gnu" - "Offload::Linker", inputs: ["[[OUTPUT]]"], output: "a.out"

// RUN: %clang -### -target x86_64-linux-gnu -nocudalib -ccc-print-bindings -fgpu-rdc \
// RUN:        --offload-new-driver --offload-arch=sm_35 --offload-arch=sm_70 %s 2>&1 \
// RUN: | FileCheck -check-prefix BINDINGS-DEVICE %s

// BINDINGS-DEVICE: # "nvptx64-nvidia-cuda" - "clang", inputs: ["[[INPUT:.+]]"], output: "[[PTX:.+]]"
// BINDINGS-DEVICE: # "nvptx64-nvidia-cuda" - "NVPTX::Assembler", inputs: ["[[PTX]]"], output: "[[CUBIN:.+]]"

// RUN: %clang -### -target x86_64-linux-gnu -nocudalib --cuda-feature=+ptx61 --offload-arch=sm_70 --cuda-path=%S/Inputs/CUDA_111/usr/local/cuda %s 2>&1 | FileCheck -check-prefix MANUAL-FEATURE %s
// MANUAL-FEATURE: -cc1{{.*}}-target-feature{{.*}}+ptx61

// RUN: %clang -### -target x86_64-linux-gnu -nocudalib -ccc-print-bindings --offload-link %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE-LINK %s

// DEVICE-LINK: "x86_64-unknown-linux-gnu" - "Offload::Linker", inputs: ["[[INPUT:.+]]"], output: "a.out"

// RUN: %clang -### -target x86_64-linux-gnu -nocudalib --offload-new-driver \
// RUN:   --offload-arch=sm_35 --offload-arch=sm_70 --cuda-path=%S/Inputs/CUDA_111/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefix GPU-BINARY %s

// GPU-BINARY: fatbinary{{.*}}"--create" "{{.*}}.fatbin"
// GPU-BINARY: -cc1{{.*}}-fcuda-include-gpubinary" "{{.*}}.fatbin"
